#include "hip/hip_runtime.h"
#include <>
#include <cstdint>



__global__
void goGPU(float * r
	   ,float const * __restrict__ x
	   ,float const * __restrict__ y
	   ,float const * __restrict__ z
	   ,float const * __restrict__ w
	   ,uint32_t n
	   ) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i>=n) return;
	   
  r[i] = x[i]*y[i] + z[i]*w[i];
}


int main() {

  uint32_t N = 1024;

  float x[1024];
  float * x_d = nullptr;
  hipMalloc(&x_d, sizeof(x));


  float r[1024];
  float * r_d = nullptr;
  hipMalloc(&r_d, sizeof(r));

  
  hipMemcpy(x_d, x, sizeof(r), hipMemcpyHostToDevice);

  goGPU<<<4,256>>>(r_d,x_d,y_d,z_d,w_d,N);
  
  hipMemcpy(r, r_d, sizeof(r), hipMemcpyDeviceToHost);

  for ( e : r)
    std::cout << e ' ';
  std::endl;


  
}
